//#include "pch.h"

#define _SILENCE_ALL_CXX23_DEPRECATION_WARNINGS
#define _AEF_WILL_USE_CUDA_HEADERS
#include "aef/matrix_utils.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hipsolver.h>
#include "cuda_utils.h"
#include "cusolver_utils.h"

namespace aef {
    // cuda and cusolver props
    namespace {
        bool init = false;
        // selected device ID
        int devID = -1;
        // stream
        hipStream_t cu_stream;
        hipsolverHandle_t cu_handle = 0;
        hipDeviceProp_t deviceProps;
        // device matrix pointer --> used both for input hermitian matrix and for evec output
        hipDoubleComplex* d_A;
        // device eigenvalues pointer --> note: this is real bec
        double *d_W;
        // device info ptr
        int *d_info = nullptr;

        //
        int saved_n = -1;
    };


    bool init_cuda(int argc, const char **argv) {
        if (init) {
            return true;
        }
        devID = cuda::findCudaDevice(argc, argv);
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
        checkCudaErrors(hipSetDevice(devID));
        checkCudaErrors(hipsolverDnCreate(&cu_handle));
        checkCudaErrors(hipStreamCreateWithFlags(&cu_stream, hipStreamNonBlocking));
        checkCudaErrors(hipsolverSetStream(cu_handle, cu_stream));
        saved_n = -1;
        init = true;
        return true;
    }

    bool shutdown_cuda() {
        if (!init) {
            return true;
        }
        hipStreamSynchronize(cu_stream);
        if (d_A) {
            checkCudaErrors(hipFreeAsync(d_A, cu_stream));
            d_A = nullptr;
        }

        if (d_W) {
            checkCudaErrors(hipFreeAsync(d_W, cu_stream));
            d_W = nullptr;
        }

        if (d_info) {
            checkCudaErrors(hipFreeAsync(d_info, cu_stream));
            d_info = nullptr;
        }

        checkCudaErrors(hipStreamSynchronize(cu_stream));
        checkCudaErrors(hipsolverDnDestroy(cu_handle));
        cu_handle = nullptr;
        checkCudaErrors(hipStreamDestroy(cu_stream));
        cu_stream = nullptr;
        checkCudaErrors(hipDeviceReset());
        init = false;
        return true;
    }

    bool is_cuda_initialized() {
        return init;
    }

    void mat_init(hipStream_t stream) {
        auto status = hipsolverDnCreate(&cu_handle);
        cu_stream = stream;
        CUSOLVER_CHECK(hipsolverSetStream(cu_handle, stream));
        init = true;
    }

    void cuda_resize(int n) {
        assert(n >= 0);
        if (n == saved_n) {
            return;
        }
        const size_t szA = sizeof(hipDoubleComplex) * n * n;
        const size_t szW = sizeof(double) * n;

        if (d_A) {
            checkCudaErrors(hipFreeAsync(d_A, cu_stream));
            d_A = nullptr;
        }

        if (d_W) {
            checkCudaErrors(hipFreeAsync(d_W, cu_stream));
            d_W = nullptr;
        }

        if (d_info) {
            checkCudaErrors(hipFreeAsync(d_info, cu_stream));
            d_info = nullptr;
        }

        if (n == 0) {
            // don't bother allocating zero-sized arrays
            saved_n = n;
            return;
        }
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void**>(&d_A), szA, cu_stream));
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void**>(&d_W), szW, cu_stream));
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void**>(&d_info), sizeof(int), cu_stream));

        checkCudaErrors(hipStreamSynchronize(cu_stream));
        saved_n = n;
    }

    void log_dev_props_info(std::ostream& out) {
        //deviceProps.
    }


    void diagonalize(Eigen::MatrixXcd& mat, Eigen::VectorXcd& evals, Eigen::MatrixXcd& evecs) {
        const int rows = (int)mat.rows();

        if (rows <= 0) {
            // don't do work on a zero-sized matrix
            return;
        }

        std::cout << "[Cuda-based diagonalizer] Diagonalize called" << std::endl;

        if (rows > saved_n || saved_n <= 0) {
            cuda_resize(rows);
        }

        const size_t mat_size = sizeof(hipDoubleComplex) * mat.size();
        const size_t ws_size = sizeof(hipDoubleComplex) * rows;
        const hipDoubleComplex *data = reinterpret_cast<hipDoubleComplex*>(mat.data());
        hipDoubleComplex *pW = reinterpret_cast<hipDoubleComplex*>(evals.data());
        hipDoubleComplex *pV = reinterpret_cast<hipDoubleComplex*>(evecs.data());
        int info = 0;

        // upload to GPU
        checkCudaErrors(hipMemcpyAsync(d_A, data, mat_size, hipMemcpyHostToDevice, cu_stream));
        std::cout << "[Cuda-based diagonalizer] data uploaded to gpu" << std::endl;
        // allocate workspace: first query how large it needs to be, then allocate
        const auto jobz = HIPSOLVER_EIG_MODE_VECTOR;
        const auto uplo = HIPBLAS_FILL_MODE_UPPER;
        int lwork;
        checkCudaErrors(hipsolverDnZheevd_bufferSize(cu_handle, jobz, uplo, rows, d_A, rows, d_W, &lwork));
        std::cout << "[Cuda-based diagonalizer] zheev work size will be " << lwork * sizeof(hipDoubleComplex)<< " bytes" << std::endl;
        hipDoubleComplex* d_Work;
        checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_Work), lwork * sizeof(hipDoubleComplex)));
        std::cout << "[Cuda-based diagonalizer] allocated work space on gpu" << std::endl;
        
        // call cusolvers ZHEEV, then copy data back to CPU ram
        auto status = (hipsolverDnZheevd(cu_handle, jobz, uplo, rows, d_A, rows, d_W, d_Work, lwork, d_info));
        std::cout << "[Cuda-based diagonalizer] queued zheev execution" << std::endl;
        checkCudaErrors(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, cu_stream));
        std::cout << "[Cuda-based diagonalizer] scheduled zheev info output to be copied back to host" << std::endl;
        if (info != 0 || status != HIPSOLVER_STATUS_SUCCESS) {
            // errcode
            hipStreamSynchronize(cu_stream);
            std::cout << "cuSOLVER ZHEEV execution failed in " __FILE__ " at line # " << 164 << " info is " << info << std::endl;
            checkCudaErrors(status);
        }

        checkCudaErrors(hipMemcpyAsync(pV, d_A, mat_size, hipMemcpyDeviceToHost, cu_stream));
        checkCudaErrors(hipMemcpyAsync(pW, d_W, ws_size , hipMemcpyDeviceToHost, cu_stream));
        std::cout << "[Cuda-based diagonalizer] scheduled for data to be copied back to host" << std::endl;
        // wait for all operations to complete
        checkCudaErrors(hipStreamSynchronize(cu_stream));
        std::cout << "[Cuda-based diagonalizer] diagonalizaion has completed execution" << std::endl;
        
        //
        if (info != 0 || status != HIPSOLVER_STATUS_SUCCESS) {
            // errcode
            std::cout << "cuSOLVER ZHEEV execution failed in " __FILE__ " at line # " << 164 << " info is " << info << std::endl;
            checkCudaErrors(status);
        }
    }
}