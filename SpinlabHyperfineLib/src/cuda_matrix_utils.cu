//#include "pch.h"

#define _SILENCE_ALL_CXX23_DEPRECATION_WARNINGS
#define _AEF_WILL_USE_CUDA_HEADERS
#include "aef/matrix_utils.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hipsolver.h>
#include "cuda_utils.h"
#include "cusolver_utils.h"

namespace aef {
    // cuda and cusolver props
    namespace {
        bool init = false;
        // selected device ID
        int devID = -1;
        // stream
        hipStream_t cu_stream;
        hipsolverHandle_t cu_handle = 0;
        hipDeviceProp_t deviceProps;
        // device matrix pointer --> used both for input hermitian matrix and for evec output
        hipDoubleComplex* d_A;
        // device eigenvalues pointer --> note: this is real bec
        double *d_W;
        int lwork = 0;
        hipDoubleComplex* d_Work = nullptr;
        // host eigenvalues --> need this because 
        std::vector<double> h_W;
        // device info ptr
        int *d_info = nullptr;

        //
        int saved_n = -1;
    };


    bool init_cuda(int argc, const char **argv) {
        if (init) {
            return true;
        }
        devID = cuda::findCudaDevice(argc, argv);
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
        checkCudaErrors(hipSetDevice(devID));
        checkCudaErrors(hipsolverDnCreate(&cu_handle));
        checkCudaErrors(hipStreamCreateWithFlags(&cu_stream, hipStreamNonBlocking));
        checkCudaErrors(hipsolverSetStream(cu_handle, cu_stream));
        saved_n = -1;
        init = true;
        return true;
    }

    bool shutdown_cuda() {
        if (!init) {
            return true;
        }
        hipStreamSynchronize(cu_stream);
        if (d_A) {
            checkCudaErrors(hipFreeAsync(d_A, cu_stream));
            d_A = nullptr;
        }

        if (d_W) {
            checkCudaErrors(hipFreeAsync(d_W, cu_stream));
            d_W = nullptr;
        }

        if (d_info) {
            checkCudaErrors(hipFreeAsync(d_info, cu_stream));
            d_info = nullptr;
        }

        if (d_Work) {
            checkCudaErrors(hipFreeAsync(d_Work, cu_stream));
            d_Work = nullptr;
        }

        checkCudaErrors(hipStreamSynchronize(cu_stream));
        checkCudaErrors(hipsolverDnDestroy(cu_handle));
        cu_handle = nullptr;
        checkCudaErrors(hipStreamDestroy(cu_stream));
        cu_stream = nullptr;
        checkCudaErrors(hipDeviceReset());
        init = false;
        return true;
    }

    bool is_cuda_initialized() {
        return init;
    }

    void mat_init(hipStream_t stream) {
        auto status = hipsolverDnCreate(&cu_handle);
        cu_stream = stream;
        CUSOLVER_CHECK(hipsolverSetStream(cu_handle, stream));
        init = true;
    }

    void cuda_resize(int n) {
        assert(n >= 0);
        if (n == saved_n) {
            return;
        }

        assert(init);

        std::cout << "[Cuda-based diagonalizer] Resizing from " << saved_n << " rows to " << n << " rows." << std::endl;
        const size_t szA = sizeof(hipDoubleComplex) * n * n;
        const size_t szW = sizeof(double) * n;

        if (d_A) {
            checkCudaErrors(hipFreeAsync(d_A, cu_stream));
            d_A = nullptr;
        }

        if (d_W) {
            checkCudaErrors(hipFreeAsync(d_W, cu_stream));
            d_W = nullptr;
        }

        if (d_info) {
            checkCudaErrors(hipFreeAsync(d_info, cu_stream));
            d_info = nullptr;
        }

        if (n == 0) {
            // don't bother allocating zero-sized arrays
            h_W.resize(0);
            saved_n = n;
            return;
        }
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void**>(&d_A), szA, cu_stream));
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void**>(&d_W), szW, cu_stream));
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void**>(&d_info), sizeof(int), cu_stream));

        // allocate host-side eigenvalue buffer
        h_W.reserve(n);
        // pre-allocate workspace: first query how large it needs to be, then allocate
        const auto jobz = HIPSOLVER_EIG_MODE_VECTOR;
        const auto uplo = HIPBLAS_FILL_MODE_UPPER;
        checkCudaErrors(hipsolverDnZheevd_bufferSize(cu_handle, jobz, uplo, n, d_A, n, d_W, &lwork));
        std::cout << "[Cuda-based diagonalizer] zheev work size will be " << lwork * sizeof(hipDoubleComplex) << " bytes" << std::endl;
        checkCudaErrors(hipMallocAsync(reinterpret_cast<void**>(&d_Work), lwork * sizeof(hipDoubleComplex), cu_stream));

        checkCudaErrors(hipStreamSynchronize(cu_stream));
        saved_n = n;
    }

    void log_dev_props_info(std::ostream& out) {
        //deviceProps.
    }


    void diagonalize(Eigen::MatrixXcd& mat, Eigen::VectorXcd& evals, Eigen::MatrixXcd& evecs) {
        const int rows = (int)mat.rows();

        if (rows <= 0) {
            // don't do work on a zero-sized matrix
            return;
        }

        std::cout << "[Cuda-based diagonalizer] Diagonalize called" << std::endl;

        if (rows > saved_n || saved_n <= 0) {
            std::cout << "[Cuda-based diagonalizer] Automatically resizing from " << saved_n << " rows to " << rows << " rows." << std::endl;
            cuda_resize(rows);
        }

        const size_t mat_size = sizeof(hipDoubleComplex) * mat.size();
        const size_t ws_size = sizeof(double) * rows;
        const hipDoubleComplex *data = reinterpret_cast<hipDoubleComplex*>(mat.data());
        double* pW = h_W.data();//reinterpret_cast<hipDoubleComplex*>(evals.data());
        hipDoubleComplex *pV = reinterpret_cast<hipDoubleComplex*>(evecs.data());
        int info = 0;

        // upload to GPU
        checkCudaErrors(hipMemcpyAsync(d_A, data, mat_size, hipMemcpyHostToDevice, cu_stream));
        std::cout << "[Cuda-based diagonalizer] data uploaded to gpu" << std::endl;
        // check workspace buffer size is large enough
        const auto jobz = HIPSOLVER_EIG_MODE_VECTOR;
        const auto uplo = HIPBLAS_FILL_MODE_UPPER;
        int job_lwork = 0;
        checkCudaErrors(hipsolverDnZheevd_bufferSize(cu_handle, jobz, uplo, rows, d_A, rows, d_W, &job_lwork));
        // reallocate if necessary
        if (job_lwork > lwork) {
            std::cout << "[Cuda-based diagonalizer] need to reallocate zheev work space, "
                "old size was " << lwork * sizeof(hipDoubleComplex) << " bytes, new size will be " 
                << job_lwork * sizeof(hipDoubleComplex) << " bytes" << std::endl;
            checkCudaErrors(hipFree(d_Work));
            checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_Work), lwork * sizeof(hipDoubleComplex)));
            std::cout << "[Cuda-based diagonalizer] allocated new work space on gpu" << std::endl;
        } else {
            std::cout << "[Cuda-based diagonalizer] using pre-allocated workspace of " << lwork * sizeof(hipDoubleComplex) << " bytes." << std::endl;
        }
        // call cusolvers ZHEEV, then copy data back to CPU ram
        auto status = (hipsolverDnZheevd(cu_handle, jobz, uplo, rows, d_A, rows, d_W, d_Work, lwork, d_info));
        std::cout << "[Cuda-based diagonalizer] queued zheev execution" << std::endl;
        checkCudaErrors(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, cu_stream));
        std::cout << "[Cuda-based diagonalizer] scheduled zheev info output to be copied back to host" << std::endl;
        if (info != 0 || status != HIPSOLVER_STATUS_SUCCESS) {
            // errcode
            hipStreamSynchronize(cu_stream);
            std::cout << "cuSOLVER ZHEEV execution failed in " __FILE__ " at line # " << 164 << " info is " << info << std::endl;
            checkCudaErrors(status);
        }

        checkCudaErrors(hipMemcpyAsync(pV, d_A, mat_size, hipMemcpyDeviceToHost, cu_stream));
        checkCudaErrors(hipMemcpyAsync(pW, d_W, ws_size , hipMemcpyDeviceToHost, cu_stream));
        std::cout << "[Cuda-based diagonalizer] scheduled for data to be copied back to host" << std::endl;
        // wait for all operations to complete
        checkCudaErrors(hipStreamSynchronize(cu_stream));
        std::cout << "[Cuda-based diagonalizer] diagonalizaion has completed execution" << std::endl;

        // copy from host memory to eigenvalue vector
        // this is necessary because evals is a dcomplex vector
        // but CUDA outputs a real double vector.
        //std::copy(h_W.begin(), h_W.end(), evals.data());
        for (int i = 0; i < evals.size(); i++) {
            evals(i) = h_W[i];
        }
        std::cout << "[Cuda-based diagonalizer] fixed up eigenvalue vector" << std::endl;

        //
        if (info != 0 || status != HIPSOLVER_STATUS_SUCCESS) {
            // errcode
            std::cout << "cuSOLVER ZHEEV execution failed in " __FILE__ " at line # " << 164 << " info is " << info << std::endl;
            checkCudaErrors(status);
        }
    }
}